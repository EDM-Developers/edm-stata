#include "hip/hip_runtime.h"
#include "lp_distance.cuh"

#define divup(a, b) (((a) + (b)-1) / (b))

template<typename T>
__device__ constexpr T getMissingConstant()
{
  return 1.0e+100; // double
}

template<>
__device__ float getMissingConstant()
{
  return 1.0e+30;
}

template<typename T>
__device__ inline T getValue(T value)
{
  return value;
}

template<>
__device__ inline float getValue(float value)
{
  return isinf(value) ? getMissingConstant<float>() : value;
}

template<typename T, int BLOCK_DIM_X, int BLOCK_DIM_Y>
__device__ T reduceEacts(int tid, T* smem)
{
  constexpr unsigned int BLK_SIZE = BLOCK_DIM_X * BLOCK_DIM_Y;

  if (BLK_SIZE > 128) {
    if (tid < 128) {
      smem[tid] += smem[tid + 128];
    }
    __syncthreads();
  }
  if (BLK_SIZE > 64) {
    if (tid < 64) {
      smem[tid] += smem[tid + 64];
    }
    __syncthreads();
  }
  if (BLK_SIZE > 32) {
    if (tid < 32) {
      smem[tid] += smem[tid + 32];
    }
    __syncthreads();
  }

  T retVal = smem[tid];
  __syncthreads();

  return retVal;
}

template<typename T, bool isDMAE, int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void lpDistances(char* const valids, T* const distances, const int numPredictions, const bool isPanelMode,
                            const double idw, const double missingDistance, const int eacts, const int numLibraryPoints,
                            const T* mData, const int* mPanelIds, const T* mpData, const int* mpPanelIds,
                            const char* mopts)
{
  const T MISSING = getMissingConstant<T>();

  const int p = blockIdx.y; // nth prediction

  if (p < numPredictions) {
    __shared__ T dists[BLOCK_DIM_X * BLOCK_DIM_Y];
    __shared__ bool markers[BLOCK_DIM_X * BLOCK_DIM_Y];

    const bool isZero = (missingDistance == 0);
    const T* predsMp = mpData + p * eacts;

    const int tid = BLOCK_DIM_X * threadIdx.y + threadIdx.x;
    const int nob = BLOCK_DIM_X * blockIdx.x + threadIdx.x;

    if (nob < numLibraryPoints) {
      const T* predsM = mData + nob * eacts;
      bool anyEAmissing = false;
      T dist_i = T(0);

      if (threadIdx.y == 0 && isPanelMode && idw > 0) {
        dist_i += (idw * (mPanelIds[nob] != mpPanelIds[p]));
      }
      for (int e = threadIdx.y; e < eacts; e += BLOCK_DIM_Y) {
        T M_ij = getValue(predsM[e]);
        T Mp_ij = getValue(predsMp[e]);
        bool mopt = mopts[e];
        bool msng = (M_ij == MISSING || Mp_ij == MISSING);
        T diffM = M_ij - Mp_ij;
        T compM = M_ij != Mp_ij;
        T distM = mopt * diffM + (1 - mopt) * compM;
        T dist_ij = msng * (1 - isZero) * missingDistance + (1 - msng) * distM;

        if (isDMAE) {
          dist_i += abs(dist_ij) / eacts;
        } else {
          dist_i += dist_ij * dist_ij;
        }
        anyEAmissing = (anyEAmissing || msng);
      }
      __syncthreads();

      dists[tid] = dist_i;
      __syncthreads();
      dist_i = reduceEacts<T, BLOCK_DIM_X, BLOCK_DIM_Y>(tid, dists);

      markers[tid] = anyEAmissing;
      __syncthreads();
      anyEAmissing = reduceEacts<bool, BLOCK_DIM_X, BLOCK_DIM_Y>(tid, markers) > 0;

      // Only first warp writes numLibraryPoints results
      if (tid < BLOCK_DIM_X) {
        anyEAmissing = anyEAmissing && isZero;

        dist_i = anyEAmissing * MISSING + (1 - anyEAmissing) * dist_i;

        bool isValid = dist_i != 0 && dist_i != MISSING;

        dist_i = (isDMAE * dist_i + (1 - isDMAE) * sqrt(dist_i));

        valids[nob + p * numLibraryPoints] = (char)isValid;
        distances[nob + p * numLibraryPoints] = dist_i;
      }
    }
  }
}

template<typename T, int BLOCK_DIM_Y>
void lpDistances(char* const valids, T* const distances, const int numPredictions, const bool isDMAE,
                 const bool isPanelMode, const double idw, const double missingDistance, const int eacts,
                 const int numLibraryPoints, const T* mData, const int* mPanelIds, const T* mpData,
                 const int* mpPanelIds, const char* mopts, const hipStream_t stream, const dim3 blocks,
                 const dim3 threads)
{
  if (isDMAE) {
    lpDistances<T, true, 32, BLOCK_DIM_Y>
      <<<blocks, threads, 0, stream>>>(valids, distances, numPredictions, isPanelMode, idw, missingDistance, eacts,
                                       numLibraryPoints, mData, mPanelIds, mpData, mpPanelIds, mopts);
  } else {
    lpDistances<T, false, 32, BLOCK_DIM_Y>
      <<<blocks, threads, 0, stream>>>(valids, distances, numPredictions, isPanelMode, idw, missingDistance, eacts,
                                       numLibraryPoints, mData, mPanelIds, mpData, mpPanelIds, mopts);
  }
}

inline unsigned int powerOf2LE(unsigned int value)
{
  value |= value >> 1;
  value |= value >> 2;
  value |= value >> 4;
  value |= value >> 8;
  value |= value >> 16;

  return value ^ (value >> 1);
}

template<typename T>
void cuLPDistances(char* const valids, T* const distances, const int numPredictions, const bool isDMAE,
                   const bool isPanelMode, const double idw, const double missingDistance, const int eacts,
                   const int numLibraryPoints, const T* mData, const int* mPanelIds, const T* mpData,
                   const int* mpPanelIds, const char* mopts, const hipStream_t stream)
{
  dim3 threads(32, powerOf2LE(eacts));

  threads.y = (threads.y > 8 ? 8 : threads.y);

  dim3 blocks(divup(numLibraryPoints, threads.x), numPredictions);

  switch (threads.y) {
    case 8:
      lpDistances<T, 8>(valids, distances, numPredictions, isDMAE, isPanelMode, idw, missingDistance, eacts,
                        numLibraryPoints, mData, mPanelIds, mpData, mpPanelIds, mopts, stream, blocks, threads);
      break;
    case 4:
      lpDistances<T, 4>(valids, distances, numPredictions, isDMAE, isPanelMode, idw, missingDistance, eacts,
                        numLibraryPoints, mData, mPanelIds, mpData, mpPanelIds, mopts, stream, blocks, threads);
      break;
    default:
      lpDistances<T, 2>(valids, distances, numPredictions, isDMAE, isPanelMode, idw, missingDistance, eacts,
                        numLibraryPoints, mData, mPanelIds, mpData, mpPanelIds, mopts, stream, blocks, threads);
      break;
  }
}

#define INSTANTIATE(T)                                                                                                 \
  template void cuLPDistances(char* const, T* const, const int, const bool, const bool, const double, const double,    \
                              const int, const int, const T*, const int*, const T*, const int*, const char*,           \
                              const hipStream_t);

INSTANTIATE(float)
INSTANTIATE(double)
